
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
using namespace std;
__global__ void BucketSort(int * bucket,int *key,int N,int range){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  bucket[i%5] = 0;
  __syncthreads();
  if(key[i] == 0){
    atomicAdd(&bucket[0],1);
  }else if(key[i] == 1){
    atomicAdd(&bucket[1],1);
  }else if(key[i] == 2){
    atomicAdd(&bucket[2],1);
  }else if(key[i] == 3){
    atomicAdd(&bucket[3],1);
  }else{
    atomicAdd(&bucket[4],1);
  }

  if(i < bucket[0]){
    key[i] = 0;
  }else if(i < bucket[0]+bucket[1]){
    key[i] = 1;
  }else if(i < bucket[0]+bucket[1]+bucket[2]){
    key[i] = 2;
  }else if(i < bucket[0]+bucket[1]+bucket[2]+bucket[3]){
    key[i] = 3;
  }else if(i < N){
    key[i] = 4;
  }

}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key,n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket,range*sizeof(int));
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  BucketSort<<<1,n>>>(bucket,key,n,range);
  hipDeviceSynchronize();
  hipFree(bucket);

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
}
